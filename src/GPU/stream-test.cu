#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <string.h>
#include "utils_cuda.h"
#include "utils_file.h"
//#include <cutil_inline.h>
#include "data.h"
#include "timer.h"

__global__ void Fir(float *d_signal_real, float *d_signal_img, const float* coeff, const int nTaps, const int nChannels, float2 *spectra)
{
	int tx = threadIdx.x + blockDim.x*blockIdx.y;
	int index = nChannels*blockIdx.x + tx;
	int i, i_coeff, i_data;
	float local_spectra_x = 0.0f;
	float local_spectra_y = 0.0f;
	
	for(int t=0;t<nTaps;t++){
	  i = t*nChannels;
	  i_coeff = i + tx;
	  i_data = i + index;
	  local_spectra_x += coeff[i_coeff]*d_signal_real[i_data];
	  local_spectra_y += coeff[i_coeff]*d_signal_img[i_data];
	}
		
	spectra[index].x=local_spectra_x;
	spectra[index].y=local_spectra_y;
	//return;
}

__global__ void Fir_SpB(float2* d_data, float* d_coeff, int nTaps, int nChannels, int yshift, float2* d_spectra) {
	int t = 0;
	int bl= blockIdx.x*nChannels;
	int ypos = blockDim.x*blockIdx.y + yshift;
	float2 ftemp1;
	ftemp1.x=0.0f;ftemp1.y=0.0f;

	for(t=ypos + threadIdx.x;t<nTaps*nChannels;t+=nChannels){
		ftemp1.x  += d_coeff[t]*d_data[bl+t].x;
		ftemp1.y  += d_coeff[t]*d_data[bl+t].y;
	}

	t=bl + ypos + threadIdx.x;
	d_spectra[t]=ftemp1;
	return;
}


void gpu_code(  float2 *data, 
				float2 *spectra, 
				float *coeff,
				const int nChannels,
				unsigned int nBlocks, 
				unsigned int filesize,
				int blocks_y, 
				int nTaps, 
				int seg_blocks){
					
bool WRITE=true;
//------------ initialize card -----------

  int devCount, device;
  hipDeviceProp_t devProp;
    
  checkCudaErrors(hipGetDeviceCount(&devCount));
  printf("\n\t\t-------------- GPU part -----------------");
  printf("\nThere are %d devices.", devCount);

	//get number of GPU available
  for (int i = 0; i < devCount; i++){
	checkCudaErrors(hipGetDeviceProperties(&devProp,i));	
	printf("\n\t Using device:\t\t\t%s\n", devProp.name);
	printf("\t Concurrent kernels:\t\t\t%i\n", devProp.concurrentKernels);
	printf("\t Async Engine Count:\t\t\t%i\n", devProp.asyncEngineCount);
	device = 0;
	// set some preferable card
	checkCudaErrors(hipSetDevice(device));
  }

	GpuTimer timer, time_memory, time_kernels;

//------------- stream setup ------------------------------------
	hipStream_t stream0, stream1, stream2, stream3;
 
	printf("\nStream creating...\t\t\t");
	timer.Start();
			checkCudaErrors(hipStreamCreate(&stream0));
			checkCudaErrors(hipStreamCreate(&stream1));
			checkCudaErrors(hipStreamCreate(&stream2));
			checkCudaErrors(hipStreamCreate(&stream3));
	timer.Stop();
	printf("done in %g ms.\n", timer.Elapsed());
	

//---------------------------------------------------------------

//------------ memory setup -------------------------------------
	float *d_coeff;
	//int seg_blocks = 10000; // each segment compute # of spectra
	int run_blocks = nBlocks - nTaps + 1; // needed blocks for run on whole host data
	int SegSize = (seg_blocks + nTaps - 1)*nChannels; //size of each segment in the buffer
	int seg_offset = seg_blocks*nChannels;
	printf("Number of spectra per block: \t%i\n", seg_blocks);
	printf("Size of segment in bytes: \t%lu\n", SegSize*sizeof(float));
	printf("Run_blocks: \t\t\t%i\n", run_blocks);
	printf("Offset: \t\t\t%i\n", seg_offset);
	printf("-----------------------------------------\n");
	
	
	//stream 0..4
	float2 *d_spectra_0, *d_data_0;
	float2 *d_spectra_1, *d_data_1;
	float2 *d_spectra_2, *d_data_2;
	float2 *d_spectra_3, *d_data_3;

	float fir_time = 0.0f;
	float fft_time = 0.0f;
	float mem_time_in = 0.0f;
	float mem_time_out = 0.0f;
	

	// grid and block size
	//int grid_0 = (int)(seg_blocks/2);
	dim3 gridSize0( seg_blocks, blocks_y, 1);
	dim3 blockSize0(nChannels/gridSize0.y, 1, 1); 
	//dim3 gridSize1( run_blocks - grid_0, blocks_y, 1);
	//dim3 blockSize1(nChannels/gridSize1.y, 1, 1); 
		
	checkCudaErrors(hipMalloc((void **) &d_spectra_0, sizeof(float2)*SegSize));
	checkCudaErrors(hipMalloc((void **) &d_spectra_1, sizeof(float2)*SegSize));
	checkCudaErrors(hipMalloc((void **) &d_spectra_2, sizeof(float2)*SegSize));
	checkCudaErrors(hipMalloc((void **) &d_spectra_3, sizeof(float2)*SegSize));
	checkCudaErrors(hipMalloc((void **) &d_coeff,   sizeof(float)*nChannels*nTaps));
	checkCudaErrors(hipMalloc((void **) &d_data_0,    sizeof(float2)*SegSize));	
	checkCudaErrors(hipMalloc((void **) &d_data_1,    sizeof(float2)*SegSize));
	checkCudaErrors(hipMalloc((void **) &d_data_2,    sizeof(float2)*SegSize));	
	checkCudaErrors(hipMalloc((void **) &d_data_3,    sizeof(float2)*SegSize));
	
	printf("\n\t\td_spectra using filesize: \t%g MB.", 4*sizeof(float2)*SegSize/1024.0/1024);
	printf("\n\t\td_coeff using filesize: \t%g MB.", sizeof(float)*SegSize/1024.0/1024);
	printf("\n\t\td_data using filesize: \t\t%g MB.", 4*sizeof(float2)*SegSize/1024.0/1024);
	printf("\n\t\t----------------------\t\t-----------");
	printf("\n\t\tTotal: \t\t\t\t%g MB.\n\n",sizeof(float)*(17.0*SegSize)/1024/1024);
	
	//coefficients copy
	checkCudaErrors(hipMemcpy(d_coeff, coeff, nChannels*nTaps*sizeof(float), hipMemcpyHostToDevice));

	// set to 0.0
	//checkCudaErrors(hipMemset(d_spectra_0, 0.0, sizeof(float2)*SegSize));
	//checkCudaErrors(hipMemset(d_spectra_1, 0.0, sizeof(float2)*SegSize));

	//Create fft Plan
	hipfftHandle plan0;
	hipfftHandle plan1;
	hipfftHandle plan2;
	hipfftHandle plan3;
	hipfftPlan1d(&plan0, nChannels, HIPFFT_C2C, seg_blocks);
	hipfftPlan1d(&plan1, nChannels, HIPFFT_C2C, seg_blocks);
	hipfftPlan1d(&plan2, nChannels, HIPFFT_C2C, seg_blocks);
	hipfftPlan1d(&plan3, nChannels, HIPFFT_C2C, seg_blocks);
	hipfftSetStream(plan0,stream0);
	hipfftSetStream(plan1,stream1);
	hipfftSetStream(plan2,stream2);
	hipfftSetStream(plan3,stream3);
	
	timer.Start();
for (int i = 0; i < run_blocks; i+=seg_blocks*4){

		checkCudaErrors(hipMemcpyAsync(d_data_0, data + i*nChannels, sizeof(float2)*SegSize, hipMemcpyHostToDevice, stream0));
		checkCudaErrors(hipMemcpyAsync(d_data_1, data + seg_offset + i*nChannels, sizeof(float2)*SegSize, hipMemcpyHostToDevice, stream1));
		checkCudaErrors(hipMemcpyAsync(d_data_2, data + 2*seg_offset + i*nChannels, sizeof(float2)*SegSize, hipMemcpyHostToDevice, stream2));
		checkCudaErrors(hipMemcpyAsync(d_data_3, data + 3*seg_offset + i*nChannels, sizeof(float2)*SegSize, hipMemcpyHostToDevice, stream3));

		Fir_SpB<<<gridSize0, blockSize0, 0, stream0>>>(d_data_0, d_coeff, nTaps, nChannels, 0, d_spectra_0);
		hipfftExecC2C(plan0, (hipfftComplex *)d_spectra_0, (hipfftComplex *)d_spectra_0, HIPFFT_FORWARD);
		checkCudaErrors(hipMemcpyAsync(spectra + i*nChannels, d_spectra_0, sizeof(float2)*SegSize, hipMemcpyDeviceToHost, stream0));

		Fir_SpB<<<gridSize0, blockSize0, 0, stream1>>>(d_data_1, d_coeff, nTaps, nChannels, 0, d_spectra_1);
		hipfftExecC2C(plan1, (hipfftComplex *)d_spectra_1, (hipfftComplex *)d_spectra_1, HIPFFT_FORWARD);
		checkCudaErrors(hipMemcpyAsync(spectra + i*nChannels + seg_offset, d_spectra_1, sizeof(float2)*SegSize, hipMemcpyDeviceToHost, stream1));
		
		Fir_SpB<<<gridSize0, blockSize0, 0, stream2>>>(d_data_2, d_coeff, nTaps, nChannels, 0, d_spectra_2);
		hipfftExecC2C(plan2, (hipfftComplex *)d_spectra_2, (hipfftComplex *)d_spectra_2, HIPFFT_FORWARD);
		checkCudaErrors(hipMemcpyAsync(spectra + i*nChannels + 2*seg_offset, d_spectra_2, sizeof(float2)*SegSize, hipMemcpyDeviceToHost, stream2));
		
		Fir_SpB<<<gridSize0, blockSize0, 0, stream3>>>(d_data_3, d_coeff, nTaps, nChannels, 0, d_spectra_3);
		hipfftExecC2C(plan3, (hipfftComplex *)d_spectra_3, (hipfftComplex *)d_spectra_3, HIPFFT_FORWARD);
		checkCudaErrors(hipMemcpyAsync(spectra + i*nChannels + 3*seg_offset, d_spectra_3, sizeof(float2)*SegSize, hipMemcpyDeviceToHost, stream3));
}

	timer.Stop();
	fir_time=timer.Elapsed();
	printf("\nDone in %g ms.\n", fir_time);

//---------------- write to file process ----------------------

	char str[200];
	sprintf(str,"GPU-stream-%s.dat",devProp.name);
	
		printf("\n Write results into file...\t");
		if (WRITE) save_time(str, nBlocks-nTaps+1, fir_time, fft_time, mem_time_in, mem_time_out, nChannels, nTaps);
		printf("\t done.\n-------------------------------------\n");


//--------------- clean-up process ----------------------------
	checkCudaErrors(hipFree(d_spectra_0));
	checkCudaErrors(hipFree(d_spectra_1));
	checkCudaErrors(hipFree(d_spectra_2));
	checkCudaErrors(hipFree(d_spectra_3));
	checkCudaErrors(hipFree(d_data_0));
	checkCudaErrors(hipFree(d_data_1));
	checkCudaErrors(hipFree(d_data_2));
	checkCudaErrors(hipFree(d_data_3));
	checkCudaErrors(hipFree(d_coeff));

	hipfftDestroy(plan0);
	hipfftDestroy(plan1);
	hipfftDestroy(plan2);
	hipfftDestroy(plan3);

	checkCudaErrors(hipStreamDestroy(stream0));
	checkCudaErrors(hipStreamDestroy(stream1));
	checkCudaErrors(hipStreamDestroy(stream2));
	checkCudaErrors(hipStreamDestroy(stream3));

}
