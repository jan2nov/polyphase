#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <string.h>
#include "utils_cuda.h"
//#include <cutil_inline.h>
#include "data.h"
#include "timer.h"

__global__ void Fir(float *d_signal_real, float *d_signal_img, const float* coeff, const int nTaps, const int nChannels, float2 *spectra)
{
	int tx = threadIdx.x + blockDim.x*blockIdx.y;
	int index = nChannels*blockIdx.x + tx;
	int i, i_coeff, i_data;
	float local_spectra_x = 0.0f;
	float local_spectra_y = 0.0f;
	
	for(int t=0;t<nTaps;t++){
	  i = t*nChannels;
	  i_coeff = i + tx;
	  i_data = i + index;
	  local_spectra_x += coeff[i_coeff]*d_signal_real[i_data];
	  local_spectra_y += coeff[i_coeff]*d_signal_img[i_data];
	}
		
	spectra[index].x=local_spectra_x;
	spectra[index].y=local_spectra_y;
	//return;
}


void gpu_code(  float *real,
				float *img, 
				float2 *spectra, 
				float *coeff,
				const int nChannels,
				unsigned int nBlocks, 
				unsigned int filesize,
				int blocks_y){
//------------ initialize card -----------

  int devCount, device;
  GpuTimer timer,timer2;
  
  checkCudaErrors(hipGetDeviceCount(&devCount));
  printf("\n\t\t-------------- GPU part -----------------");
  printf("\nThere are %d devices.", devCount);

  for (int i = 0; i < devCount -1 ; i++){
	hipDeviceProp_t devProp;
	checkCudaErrors(hipGetDeviceProperties(&devProp,i));	
	printf("\n\t Using device:\t\t\t%s\n", devProp.name);
	device = i;
	checkCudaErrors(hipSetDevice(device));
  }

//------------ memory setup -------------------------------------
	timer2.Start();
	float2 *d_spectra;
	float  *d_real, *d_img, *d_coeff;

	float run_time = -1.1f;

	//malloc
	printf("\nDevice memory allocation...\t\t");
	timer.Start();
	checkCudaErrors(hipMalloc((void **) &d_spectra, sizeof(float2)*filesize));
	checkCudaErrors(hipMalloc((void **) &d_coeff,   sizeof(float)*nChannels*nTaps));
	checkCudaErrors(hipMalloc((void **) &d_real,    sizeof(float)*filesize));
	checkCudaErrors(hipMalloc((void **) &d_img,     sizeof(float)*filesize));
	timer.Stop();
	printf("done in %g ms.", timer.Elapsed());

	// set to 0.0
	printf("\nDevice memset...\t\t\t");
	timer.Start();
	checkCudaErrors(hipMemset(d_spectra, 0.0, sizeof(float2)*filesize));
	timer.Stop();
	printf("done in %g ms.", timer.Elapsed());

	// copy data to device
	printf("\nCopy data from host to device...\t");
	timer.Start();
	checkCudaErrors(hipMemcpy(d_coeff, coeff, nChannels*nTaps*sizeof(float), hipMemcpyHostToDevice));
	//checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(C_coeff),  coeff,   sizeof(float)*nChannels*nTaps));
	checkCudaErrors(hipMemcpy(d_real, real, filesize*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_img,  img,  filesize*sizeof(float), hipMemcpyHostToDevice));
	timer.Stop();
	printf("done in %g ms.", timer.Elapsed());
//---------------------------------------------------------

//--------------- Fir ----------------------------

	//dim3 gridSize(1, (int)(nChannels*nSpectra-1)/blocks_y + 1, 1);
	dim3 gridSize( (nBlocks - nTaps + 1), blocks_y, 1);
	dim3 blockSize(nChannels/gridSize.y, 1, 1); 
	
	timer.Start();
		Fir<<<gridSize, blockSize>>>(d_real, d_img, d_coeff, nTaps, nChannels, d_spectra);
	timer.Stop();
	run_time=timer.Elapsed();
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	printf("\n\t\t------------ Kernel run -----------------");
	printf("\nFir kernel \n");
	printf("\n\n blocks \t time \t\t threads \t bandwidth \t flops");
	printf("\n%d \t\t %lf \t %i \t\t %g \t %g\n",nBlocks/12,run_time,nChannels/blocks_y, 53248.0*(nBlocks-nTaps+1)*1000/run_time, 16384.0*(nBlocks-nTaps+1)*1000.0/run_time);

//--------------- cuFFT ----------------------------
/*
	//Create fft Plan
	hipfftHandle plan;
	hipfftPlan1d(&plan, nChannels, HIPFFT_C2C, nBlocks);

	//execute plan and copy back to host
	printf("\n\ncuFFT plan...\t\t");
	timer.Start();
	hipfftExecC2C(plan, (hipfftComplex *)d_spectra, (hipfftComplex *)d_spectra, HIPFFT_FORWARD);
	timer.Stop();
	printf("done in %g ms.\n\n", timer.Elapsed());

	//Destroy the cuFFT plan
	hipfftDestroy(plan);
*/
//--------------- copy data back ----------------------------
	checkCudaErrors(hipMemcpy(spectra,d_spectra,filesize*sizeof(float2), hipMemcpyDeviceToHost));	
timer2.Stop();
printf("\nDone in %g ms.\n", timer2.Elapsed());
//--------------- clean-up process ----------------------------
	
	checkCudaErrors(hipFree(d_spectra));
	checkCudaErrors(hipFree(d_real));
	checkCudaErrors(hipFree(d_img));
	checkCudaErrors(hipFree(d_coeff));
checkCudaErrors(hipDeviceReset());
}
